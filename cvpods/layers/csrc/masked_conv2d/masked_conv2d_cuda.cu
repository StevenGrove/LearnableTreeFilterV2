#include "hip/hip_runtime.h"
#include "masked_conv2d.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <THC/THCAtomics.cuh>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CPU(x) \
  TORCH_CHECK(!x.device().is_cuda(), #x " must be a CPU tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_CUDA_INPUT(x) \
  CHECK_CUDA(x);            \
  CHECK_CONTIGUOUS(x)
#define CHECK_CPU_INPUT(x) \
  CHECK_CPU(x);            \
  CHECK_CONTIGUOUS(x)

#define THREADS_PER_BLOCK 512

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

using at::Tensor;

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 4096;
  return min(optimal_block_num, max_block_num);
}

namespace cvpods {

void MaskedIm2colForwardCUDAKernelLauncher(const Tensor bottom_data,
                                           const Tensor mask_h_idx,
                                           const Tensor mask_w_idx,
                                           Tensor top_data, const int kernel_h,
                                           const int kernel_w, const int pad_h,
                                           const int pad_w);

void MaskedCol2imForwardCUDAKernelLauncher(const Tensor bottom_data,
                                           const Tensor mask_h_idx,
                                           const Tensor mask_w_idx,
                                           Tensor top_data, const int height,
                                           const int width, const int channels);

void masked_im2col_forward_cuda(const Tensor im, const Tensor mask_h_idx,
                                const Tensor mask_w_idx, Tensor col,
                                const int kernel_h, const int kernel_w,
                                const int pad_h, const int pad_w) {
  // im: (n, ic, h, w), kernel size (kh, kw)
  // kernel: (oc, ic * kh * kw), col: (kh * kw * ic, ow * oh)
  MaskedIm2colForwardCUDAKernelLauncher(im, mask_h_idx, mask_w_idx, col,
                                        kernel_h, kernel_w, pad_h, pad_w);
}

void masked_col2im_forward_cuda(const Tensor col, const Tensor mask_h_idx,
                                const Tensor mask_w_idx, Tensor im, int height,
                                int width, int channels) {
  // im: (n, ic, h, w), kernel size (kh, kw)
  // kernel: (oc, ic * kh * kh), col: (kh * kw * ic, ow * oh)
  MaskedCol2imForwardCUDAKernelLauncher(col, mask_h_idx, mask_w_idx, im, height,
                                        width, channels);
}

void masked_im2col_forward(const Tensor im, const Tensor mask_h_idx,
                           const Tensor mask_w_idx, Tensor col,
                           const int kernel_h, const int kernel_w,
                           const int pad_h, const int pad_w) {
  if (im.device().is_cuda()) {
    CHECK_CUDA_INPUT(im);
    CHECK_CUDA_INPUT(mask_h_idx);
    CHECK_CUDA_INPUT(mask_w_idx);
    CHECK_CUDA_INPUT(col);
    masked_im2col_forward_cuda(im, mask_h_idx, mask_w_idx, col, kernel_h,
                               kernel_w, pad_h, pad_w);
  } else {
    AT_ERROR("MaskConv is not implemented on CPU");
  }
}

void masked_col2im_forward(const Tensor col, const Tensor mask_h_idx,
                           const Tensor mask_w_idx, Tensor im, int height,
                           int width, int channels) {
  if (col.device().is_cuda()) {
    CHECK_CUDA_INPUT(col);
    CHECK_CUDA_INPUT(mask_h_idx);
    CHECK_CUDA_INPUT(mask_w_idx);
    CHECK_CUDA_INPUT(im);
    masked_col2im_forward_cuda(col, mask_h_idx, mask_w_idx, im, height, width,
                               channels);
  } else {
    AT_ERROR("MaskConv is not implemented on CPU");
  }
}

template <typename scalar_t>
__global__ void MaskedIm2colForward(const int n, const scalar_t *data_im,
                                    const int height, const int width,
                                    const int kernel_h, const int kernel_w,
                                    const int pad_h, const int pad_w,
                                    const int64_t *mask_h_idx,
                                    const int64_t *mask_w_idx,
                                    const int mask_cnt, scalar_t *data_col) {
  // mask_cnt * channels
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int m_index = index % mask_cnt;
    const int h_col = mask_h_idx[m_index];
    const int w_col = mask_w_idx[m_index];
    const int c_im = index / mask_cnt;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col - pad_h;
    const int w_offset = w_col - pad_w;
    scalar_t *data_col_ptr = data_col + c_col * mask_cnt + m_index;
    for (int i = 0; i < kernel_h; ++i) {
      int h_im = h_offset + i;
      for (int j = 0; j < kernel_w; ++j) {
        int w_im = w_offset + j;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
          *data_col_ptr =
              (scalar_t)data_im[(c_im * height + h_im) * width + w_im];
        } else {
          *data_col_ptr = 0.0;
        }
        data_col_ptr += mask_cnt;
      }
    }
  }
}

template <typename scalar_t>
__global__ void MaskedCol2imForward(const int n, const scalar_t *data_col,
                                    const int height, const int width,
                                    const int channels,
                                    const int64_t *mask_h_idx,
                                    const int64_t *mask_w_idx,
                                    const int mask_cnt, scalar_t *data_im) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int m_index = index % mask_cnt;
    const int h_im = mask_h_idx[m_index];
    const int w_im = mask_w_idx[m_index];
    const int c_im = index / mask_cnt;
    // compute the start and end of the output
    data_im[(c_im * height + h_im) * width + w_im] = data_col[index];
  }
}

void MaskedIm2colForwardCUDAKernelLauncher(const Tensor bottom_data,
                                           const Tensor mask_h_idx,
                                           const Tensor mask_w_idx,
                                           Tensor top_data, const int kernel_h,
                                           const int kernel_w, const int pad_h,
                                           const int pad_w) {
  int channels = bottom_data.size(1);
  int height = bottom_data.size(2);
  int width = bottom_data.size(3);
  int mask_cnt = mask_h_idx.size(0);
  int output_size = mask_cnt * channels;

  at::cuda::CUDAGuard device_guard(bottom_data.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.scalar_type(), "MaskedIm2colLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data_ptr<scalar_t>();
        const int64_t *mask_h_idx_ = mask_h_idx.data_ptr<int64_t>();
        const int64_t *mask_w_idx_ = mask_w_idx.data_ptr<int64_t>();
        scalar_t *top_data_ = top_data.data_ptr<scalar_t>();
        MaskedIm2colForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, bottom_data_, height, width, kernel_h, kernel_w,
                pad_h, pad_w, mask_h_idx_, mask_w_idx_, mask_cnt, top_data_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void MaskedCol2imForwardCUDAKernelLauncher(
    const Tensor bottom_data, const Tensor mask_h_idx, const Tensor mask_w_idx,
    Tensor top_data, const int height, const int width, const int channels) {
  int mask_cnt = mask_h_idx.size(0);
  int output_size = mask_cnt * channels;

  at::cuda::CUDAGuard device_guard(bottom_data.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.scalar_type(), "MaskedCol2imLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data_ptr<scalar_t>();
        const int64_t *mask_h_idx_ = mask_h_idx.data_ptr<int64_t>();
        const int64_t *mask_w_idx_ = mask_w_idx.data_ptr<int64_t>();
        scalar_t *top_data_ = top_data.data_ptr<scalar_t>();

        MaskedCol2imForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, bottom_data_, height, width, channels, mask_h_idx_,
                mask_w_idx_, mask_cnt, top_data_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

} // namespace cvpods
