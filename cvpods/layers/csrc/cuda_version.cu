// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace cvpods {
int get_cudart_version() {
  return CUDART_VERSION;
}
} // namespace cvpods
